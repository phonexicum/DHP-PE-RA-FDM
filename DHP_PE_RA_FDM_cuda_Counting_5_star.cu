#include "hip/hip_runtime.h"
#include "DHP_PE_RA_FDM.h"
#include "cuda_utils.h"


// ==================================================================================================================================================
//                                                                                                                 cudakernel_Counting_5_star_insides
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_insides (double* const delta_f, const double* const f,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 1 + k % (procCoords.x_cells_num -2);
        int j = 1 + k / (procCoords.x_cells_num -2);
        
        if (j < procCoords.y_cells_num -1) {
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                    (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                    (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                 cudakernel_Counting_5_star_Memcpy_vertical_message
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_Memcpy_vertical_message (double* const to, const double* const from, const int elem_num, const int step){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; k < elem_num; k += threadNum)
        to[k] = from[k * step];
}


// ==================================================================================================================================================
//                                                                                                              cudakernel_Counting_5_star_LR_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_LR_delta_f (double* const delta_f, const double* const f, const double* const recv_message_lr,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 0;
        int j = 1 + k;

        if (j < procCoords.y_cells_num -1) {
            if (not procCoords.left) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - recv_message_lr[j]               ) -
                        (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                        (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                              cudakernel_Counting_5_star_RL_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_RL_delta_f (double* const delta_f, const double* const f, const double* const recv_message_rl,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = procCoords.x_cells_num -1;
        int j = 1 + k;

        if (j < procCoords.y_cells_num -1) {
            if (not procCoords.right) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                        (recv_message_rl[j]                  - f[j * procCoords.x_cells_num + i  ])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                        (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                              cudakernel_Counting_5_star_TD_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_TD_delta_f (double* const delta_f, const double* const f, const double* const recv_message_td,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 1 + k;
        int j = 0;

        if (i < procCoords.x_cells_num -1) {
            if (not procCoords.top) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                        (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - recv_message_td[i]               ) -
                        (f[(j+1) * procCoords.x_cells_num + i] - f[j * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                              cudakernel_Counting_5_star_BU_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_BU_delta_f (double* const delta_f, const double* const f, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 1 + k;
        int j = procCoords.y_cells_num -1;

        if (i < procCoords.x_cells_num -1) {
            if (not procCoords.bottom) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                        (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                        (recv_message_bu[i]                    - f[ j    * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                            cudakernel_Counting_5_star_TDBU_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_TDBU_delta_f (double* const delta_f, const double* const f,
    const double* const recv_message_td, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 1 + k;
        int j = 0;

        if (i < procCoords.x_cells_num -1) {
            if (not procCoords.top and not procCoords.bottom) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                        (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - recv_message_td[i]               ) -
                        (recv_message_bu[i]                    - f[j * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                            cudakernel_Counting_5_star_LRRL_delta_f
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_LRRL_delta_f (double* const delta_f, const double* const f,
    const double* const recv_message_lr, const double* const recv_message_rl,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;
    int threadNum = GRID_SIZE_IN_THREADS;

    for (int k = threadId; true; k += threadNum) {

        int i = 0;
        int j = 1 + k;

        if (j < procCoords.y_cells_num -1) {
            if (not procCoords.left and not procCoords.right) {
                delta_f[j * procCoords.x_cells_num + i] = (
                        (f[j * procCoords.x_cells_num + i  ] - recv_message_lr[j]               ) -
                        (recv_message_rl[j]                  - f[j * procCoords.x_cells_num + i])
                    ) / hx2 + (
                        (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                        (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                    ) / hy2;
            }
        } else {
            break;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                             cudakernel_Counting_5_star_nxm_corners
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_nxm_corners (double* const delta_f, const double* const f,
    const double* const recv_message_lr, const double* const recv_message_rl,
    const double* const recv_message_td, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;

    int i;
    int j;

    if (threadId == 0){
        j = 0;
        i = 0;
        if (not procCoords.top and not procCoords.left) {
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - recv_message_lr [0] ) -
                    (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - recv_message_td [0]                  ) -
                    (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    } else if (threadId == 1){
        j = 0;
        i = procCoords.x_cells_num -1;
        if (not procCoords.top and not procCoords.right){
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                    (recv_message_rl [0]                 - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - recv_message_td [procCoords.x_cells_num -1] ) -
                    (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    } else if (threadId == 2){
        j = procCoords.y_cells_num -1;
        i = 0;
        if (not procCoords.bottom and not procCoords.left){
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - recv_message_lr[procCoords.y_cells_num -1] ) -
                    (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                    (recv_message_bu [0]                   - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    } else if (threadId == 3){
        j = procCoords.y_cells_num -1;
        i = procCoords.x_cells_num -1;
        if (not procCoords.bottom and not procCoords.right){
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                    (recv_message_rl [procCoords.y_cells_num -1] - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                    (recv_message_bu [procCoords.x_cells_num -1] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                             cudakernel_Counting_5_star_nx1_corners
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_nx1_corners (double* const delta_f, const double* const f,
    const double* const recv_message_lr, const double* const recv_message_rl,
    const double* const recv_message_td, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;

    int i;
    int j;

    if (threadId == 0){
        j = 0;
        i = 0;
        if (not procCoords.top and not procCoords.bottom and not procCoords.left) {
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - recv_message_lr [0] ) -
                    (f[j * procCoords.x_cells_num + i+1] - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - recv_message_td [0]                  ) -
                    (recv_message_bu[0]                    - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    } else if (threadId == 1){
        j = 0;
        i = procCoords.x_cells_num -1;
        if (not procCoords.top and not procCoords.bottom and not procCoords.right){
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - f[j * procCoords.x_cells_num + i-1]) -
                    (recv_message_rl [0]                 - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - recv_message_td [procCoords.x_cells_num -1]) -
                    (recv_message_bu [procCoords.x_cells_num -1] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                             cudakernel_Counting_5_star_1xm_corners
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_1xm_corners (double* const delta_f, const double* const f,
    const double* const recv_message_lr, const double* const recv_message_rl,
    const double* const recv_message_td, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int threadId = THREAD_IN_GRID_ID;

    int i;
    int j;

    if (threadId == 0){
        j = 0;
        i = 0;
        if (not procCoords.left and not procCoords.right and not procCoords.top) {
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - recv_message_lr [0]                ) -
                    (recv_message_rl[0]                  - f[j * procCoords.x_cells_num + i  ])
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - recv_message_td [0]                  ) -
                    (f[(j+1) * procCoords.x_cells_num + i] - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    } else if (threadId == 1){
        j = procCoords.y_cells_num -1;
        i = 0;
        if (not procCoords.left and not procCoords.right and not procCoords.bottom){
            delta_f[j * procCoords.x_cells_num + i] = (
                    (f[j * procCoords.x_cells_num + i  ] - recv_message_lr[j]                   ) -
                    (recv_message_rl[j]                  - f[j * procCoords.x_cells_num + i  ]  )
                ) / hx2 + (
                    (f[ j    * procCoords.x_cells_num + i] - f[(j-1) * procCoords.x_cells_num + i]) -
                    (recv_message_bu [0]                   - f[ j    * procCoords.x_cells_num + i])
                ) / hy2;
        }
    }
}


// ==================================================================================================================================================
//                                                                                                             cudakernel_Counting_5_star_1xm_corners
// ==================================================================================================================================================
__global__ void cudakernel_Counting_5_star_1x1_corners (double* const delta_f, const double* const f,
    const double* const recv_message_lr, const double* const recv_message_rl,
    const double* const recv_message_td, const double* const recv_message_bu,
    const ProcComputingCoords procCoords, const double hx2, const double hy2){

    int i = 0;
    int j = 0;
    if (not procCoords.left and not procCoords.right and not procCoords.top and not procCoords.bottom){
        delta_f[j * procCoords.x_cells_num + i] = (
                (f[j * procCoords.x_cells_num + i  ] - recv_message_lr[j]                   ) -
                (recv_message_rl[j]                  - f[j * procCoords.x_cells_num + i  ]  )
            ) / hx2 + (
                (f[ j    * procCoords.x_cells_num + i] - recv_message_td[0]                   ) -
                (recv_message_bu [0]                   - f[ j    * procCoords.x_cells_num + i])
            ) / hy2;
    }

}


// ==================================================================================================================================================
//                                                                                                                DHP_PE_RA_FDM::cuda_Counting_5_star
// ==================================================================================================================================================
void DHP_PE_RA_FDM::cuda_Counting_5_star (double* const delta_f, const double* const f){

    int ret = MPI_SUCCESS;

    int dimension = (procCoords.x_cells_num -2) * (procCoords.y_cells_num -2);
    GridDistribute mesh (devProp, dimension);
    cudakernel_Counting_5_star_insides<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[0]>>> (delta_f, f, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

    // ==========================================
    // memory allocation
    // ==========================================

    if (send_message_lr == NULL)
        SAFE_CUDA(hipHostAlloc(&send_message_lr, procCoords.y_cells_num * sizeof(*send_message_lr), hipHostMallocMapped));
    if (send_message_rl == NULL)
        SAFE_CUDA(hipHostAlloc(&send_message_rl, procCoords.y_cells_num * sizeof(*send_message_rl), hipHostMallocMapped));
    if (send_message_td == NULL)
        SAFE_CUDA(hipHostAlloc(&send_message_td, procCoords.x_cells_num * sizeof(*send_message_td), hipHostMallocMapped));
    if (send_message_bu == NULL)
        SAFE_CUDA(hipHostAlloc(&send_message_bu, procCoords.x_cells_num * sizeof(*send_message_bu), hipHostMallocMapped));
    if (recv_message_lr == NULL)
        SAFE_CUDA(hipHostAlloc(&recv_message_lr, procCoords.y_cells_num * sizeof(*recv_message_lr), hipHostMallocMapped));
    if (recv_message_rl == NULL)
        SAFE_CUDA(hipHostAlloc(&recv_message_rl, procCoords.y_cells_num * sizeof(*recv_message_rl), hipHostMallocMapped));
    if (recv_message_td == NULL)
        SAFE_CUDA(hipHostAlloc(&recv_message_td, procCoords.x_cells_num * sizeof(*recv_message_td), hipHostMallocMapped));
    if (recv_message_bu == NULL)
        SAFE_CUDA(hipHostAlloc(&recv_message_bu, procCoords.x_cells_num * sizeof(*recv_message_bu), hipHostMallocMapped));

    // ==========================================
    // initialize send buffers
    // ==========================================

    mesh = GridDistribute (devProp, procCoords.y_cells_num);

    // left -> right
    cudakernel_Counting_5_star_Memcpy_vertical_message<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[1]>>> (
        send_message_lr, f + procCoords.x_cells_num -1, procCoords.y_cells_num, procCoords.x_cells_num); CUDA_CHECK_LAST_ERROR;
    // right -> left
    cudakernel_Counting_5_star_Memcpy_vertical_message<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[2]>>> (
        send_message_rl, f, procCoords.y_cells_num, procCoords.x_cells_num); CUDA_CHECK_LAST_ERROR;
    // top -> down
    SAFE_CUDA(hipMemcpyAsync(send_message_td, f + (procCoords.y_cells_num -1) * procCoords.x_cells_num, procCoords.x_cells_num * sizeof(*f),
        hipMemcpyDeviceToHost, cudaStreams[3]));
    // bottom -> up
    SAFE_CUDA(hipMemcpyAsync(send_message_bu, f, procCoords.x_cells_num * sizeof(*f), hipMemcpyDeviceToHost, cudaStreams[4]));


    int send_amount = 0;
    int recv_amount = 0;

    // ==========================================
    // send messages
    // ==========================================

    // left -> right
    if (not procCoords.right){

        hipStreamSynchronize(cudaStreams[1]); CUDA_CHECK_LAST_ERROR;
        ret = MPI_Isend(
            send_message_lr,                            // void* buffer
            procCoords.y_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank +1,                         // int dest
            DHP_PE_RA_FDM::StarLeftRight,               // int tag
            procParams.comm,                            // MPI_Comm comm
            &(send_reqs_5_star[send_amount])            // MPI_Request *request
        );
        send_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error sending message from left to right.");
    }
    // right -> left
    if (not procCoords.left){

        hipStreamSynchronize(cudaStreams[2]); CUDA_CHECK_LAST_ERROR;
        ret = MPI_Isend(
            send_message_rl,                            // void* buffer
            procCoords.y_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank -1,                         // int dest
            DHP_PE_RA_FDM::StarRightLeft,               // int tag
            procParams.comm,                            // MPI_Comm comm
            &(send_reqs_5_star[send_amount])            // MPI_Request *request
        );
        send_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error sending message from right to left.");
    }
    // top -> down
    if (not procCoords.bottom){

        hipStreamSynchronize(cudaStreams[3]); CUDA_CHECK_LAST_ERROR;
        ret = MPI_Isend(
            send_message_td,                            // void* buffer
            procCoords.x_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank + procCoords.x_proc_num,    // int dest
            DHP_PE_RA_FDM::StarTopDown,                 // int tag
            procParams.comm,                            // MPI_Comm comm
            &(send_reqs_5_star[send_amount])            // MPI_Request *request
        );
        send_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error sending message top -> down.");
    }
    // bottom -> up
    if (not procCoords.top){

        hipStreamSynchronize(cudaStreams[4]); CUDA_CHECK_LAST_ERROR;
        ret = MPI_Isend(
            send_message_bu,                            // void* buffer
            procCoords.x_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank - procCoords.x_proc_num,    // int dest
            DHP_PE_RA_FDM::StarBottomUp,                // int tag
            procParams.comm,                            // MPI_Comm comm
            &(send_reqs_5_star[send_amount])            // MPI_Request *request
        );
        send_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error sending message bottom -> up.");
    }

    // ==========================================
    // receive messages
    // ==========================================

    // left -> right
    if (not procCoords.left){

        ret = MPI_Irecv(
            recv_message_lr,                            // void *buf
            procCoords.y_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank -1,                         // int source
            DHP_PE_RA_FDM::StarLeftRight,               // int tag
            procParams.comm,                            // MPI_Comm comm
            &(recv_reqs_5_star[recv_amount])            // MPI_Request *request
        );
        recv_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error receiving message from left to right.");
    }
    // right -> left
    if (not procCoords.right){

        ret = MPI_Irecv(
            recv_message_rl,                            // void *buf
            procCoords.y_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank +1,                         // int source
            DHP_PE_RA_FDM::StarRightLeft,               // int tag
            procParams.comm,                            // MPI_Comm comm
            &(recv_reqs_5_star[recv_amount])            // MPI_Request *request
        );
        recv_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error receiving message from right to left.");
    }
    // top -> down
    if (not procCoords.top){

        ret = MPI_Irecv(
            recv_message_td,                            // void *buf
            procCoords.x_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank - procCoords.x_proc_num,    // int source
            DHP_PE_RA_FDM::StarTopDown,                 // int tag
            procParams.comm,                            // MPI_Comm comm
            &(recv_reqs_5_star[recv_amount])            // MPI_Request *request
        );
        recv_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error receiving message top -> down.");
    }
    // bottom -> up
    if (not procCoords.bottom){

        ret = MPI_Irecv(
            recv_message_bu,                            // void *buf
            procCoords.x_cells_num,                     // int count
            MPI_DOUBLE,                                 // MPI_Datatype datatype
            procParams.rank + procCoords.x_proc_num,    // int source
            DHP_PE_RA_FDM::StarBottomUp,                // int tag
            procParams.comm,                            // MPI_Comm comm
            &(recv_reqs_5_star[recv_amount])            // MPI_Request *request
        );
        recv_amount++;

        if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error receiving message bottom -> up.");
    }

    // ==========================================
    // wait receiving all messages
    // ==========================================

    ret = MPI_Waitall(
        recv_amount,        // int count,
        recv_reqs_5_star,   // MPI_Request array_of_requests[],
        MPI_STATUS_IGNORE   // MPI_Status array_of_statuses[]
    );

    if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error waiting for recv's in Counting_5_star.");

    // ==========================================
    // process received messages
    // ==========================================

    // Counting squared regions n x m, where n > 1 and m > 1
    if (procCoords.x_cells_num > 1 and procCoords.y_cells_num > 1)
    {
        // left -> right
        mesh = GridDistribute (devProp, procCoords.y_cells_num -2);
        cudakernel_Counting_5_star_LR_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[5]>>> (
            delta_f, f, recv_message_lr, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // right -> left
        mesh = GridDistribute (devProp, procCoords.y_cells_num -2);
        cudakernel_Counting_5_star_RL_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[6]>>> (
            delta_f, f, recv_message_rl, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // top -> down
        mesh = GridDistribute (devProp, procCoords.x_cells_num -2);
        cudakernel_Counting_5_star_TD_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[7]>>> (
            delta_f, f, recv_message_td, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // bottom -> up
        mesh = GridDistribute (devProp, procCoords.x_cells_num -2);
        cudakernel_Counting_5_star_BU_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[8]>>> (
            delta_f, f, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // ==========================================
        // Counting delta_f's corners
        // ==========================================

        cudakernel_Counting_5_star_nxm_corners<<<dim3(1), dim3(4), 0, cudaStreams[9]>>> (
            delta_f, f, recv_message_lr, recv_message_rl, recv_message_td, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

    } else if (procCoords.x_cells_num > 1 and procCoords.y_cells_num == 1){
        // Counting regions n x 1, where n > 1

        // top -> down
        // bottom -> up
        mesh = GridDistribute (devProp, procCoords.x_cells_num -2);
        cudakernel_Counting_5_star_TDBU_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[5]>>> (
            delta_f, f, recv_message_td, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // ==========================================
        // Counting delta_f's corners
        // ==========================================

        cudakernel_Counting_5_star_nx1_corners<<<dim3(1), dim3(2), 0, cudaStreams[6]>>> (
            delta_f, f, recv_message_lr, recv_message_rl, recv_message_td, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

    } else if (procCoords.x_cells_num == 1 and procCoords.y_cells_num > 1){
        // Counting regions 1 x m, where m > 1

        // left -> right
        // right -> left
        mesh = GridDistribute (devProp, procCoords.y_cells_num -2);
        cudakernel_Counting_5_star_TDBU_delta_f<<<mesh.gridDim, mesh.blockDim, 0, cudaStreams[5]>>> (
            delta_f, f, recv_message_lr, recv_message_rl, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

        // ==========================================
        // Counting delta_f's corners
        // ==========================================

        cudakernel_Counting_5_star_1xm_corners<<<dim3(1), dim3(2), 0, cudaStreams[6]>>> (
            delta_f, f, recv_message_lr, recv_message_rl, recv_message_td, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;

    } else if (procCoords.x_cells_num == 1 and procCoords.y_cells_num == 1){
        // Counting regions 1 x 1

        cudakernel_Counting_5_star_1x1_corners<<<dim3(1), dim3(1), 0, cudaStreams[5]>>> (
            delta_f, f, recv_message_lr, recv_message_rl, recv_message_td, recv_message_bu, procCoords, hx2, hy2); CUDA_CHECK_LAST_ERROR;
    }

    // ==========================================
    // wait sending all messages
    // ==========================================

    ret = MPI_Waitall(
        send_amount,        // int count,
        send_reqs_5_star,   // MPI_Request array_of_requests[],
        MPI_STATUS_IGNORE   // MPI_Status array_of_statuses[]
    );

    if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error waiting for sends after previous Counting_5_star.");

    cudaAllStreamsSynchronize(0, 9);
}
