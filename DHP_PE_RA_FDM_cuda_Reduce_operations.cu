#include "hip/hip_runtime.h"
#include <utility>

using std::pair;
using std::swap;

#include "DHP_PE_RA_FDM.h"


// ==================================================================================================================================================
//                                                                                                                    cudakernel_PrepareScalarProduct
// ==================================================================================================================================================
__global__ void cudakernel_PrepareScalarProduct (double* const cuda_sum_aggr_arr, const double* const f1, const double* const f2,
    const int arr_size, const int hx, const int hy){

    int threadId = THREAD_IN_GRID_ID;

    if (threadId < arr_size)
        cuda_sum_aggr_arr[threadId] = f1[threadId] * f2[threadId] * hx * hy;
}


// ==================================================================================================================================================
//                                                                                                                              cudakernel_ComputeSum
// ==================================================================================================================================================
// 
// This cuda kernel summarize `blockSize` values from function f corresponding for current block and stores the result into
//      `cuda_sum_aggr_arr[blockLinearId]`
// 
__global__ void cudakernel_ComputeSum (double* const cuda_sum_aggr_arr, const double* const f, const int arr_size){

    const int blockSize = BLOCK_SIZE;
    int thisBlockSize;
    
    extern __shared__ double data []; // shared memory in amount of 1024 doubles (1024=maxThreadsPerBlock)

    int threadLinearBlockId = THREAD_IN_BLOCK_ID;
    int blockLinearId = BLOCK_IN_GRID_ID;

    // Cut off redundant blocks
    if (blockLinearId < (arr_size -1) / blockSize +1){

        bool lastBlock = blockLinearId == blockSize -1;
        if (lastBlock)
            thisBlockSize = arr_size % blockSize;
        else
            thisBlockSize = blockSize;

        if (not lastBlock or (lastBlock and (threadLinearBlockId < thisBlockSize)) ){

            data[threadLinearBlockId] = f[threadLinearBlockId + blockLinearId * blockSize];
            __syncthreads ();

            for (int s = 1; s < thisBlockSize; s *= 2){
                if (threadLinearBlockId % (2*s) == 0 and threadLinearBlockId + s < thisBlockSize){
                    data[threadLinearBlockId] += data[threadLinearBlockId + s];
                }
                __syncthreads ();
            }

            if (threadLinearBlockId == 0)
                cuda_sum_aggr_arr[blockLinearId] = data[0];
        }
    }
}


// ==================================================================================================================================================
//                                                                                                         DHP_PE_RA_FDM::cuda_ComputingScalarProduct
// ==================================================================================================================================================
double DHP_PE_RA_FDM::cuda_ComputingScalarProduct(const double* const f1, const double* const f2){

    if (cuda_sum_aggr_arr1 == NULL)
        SAFE_CUDA(hipMalloc(&cuda_sum_aggr_arr1, procCoords.x_cells_num * procCoords.y_cells_num * sizeof(*cuda_sum_aggr_arr1)));
    if (cuda_sum_aggr_arr2 == NULL)
        SAFE_CUDA(hipMalloc(&cuda_sum_aggr_arr2, procCoords.x_cells_num * procCoords.y_cells_num * sizeof(*cuda_sum_aggr_arr2)));


    int dimension = procCoords.x_cells_num * procCoords.y_cells_num;
    pair<dim3, dim3> mesh = GridDistribute(dimension);
    cudakernel_PrepareScalarProduct<<<mesh.first, mesh.second>>> (cuda_sum_aggr_arr1, f1, f2, dimension, hx, hy);

    while (true){
        cudakernel_ComputeSum<<<mesh.first, mesh.second, devProp.maxThreadsPerBlock * sizeof(*cuda_sum_aggr_arr1)>>> (cuda_sum_aggr_arr2, cuda_sum_aggr_arr1, dimension);

        if (dimension == 1) break;
        
        dimension = mesh.second.x * mesh.second.y * mesh.second.z; // number of blocks
        mesh = GridDistribute(dimension);
        swap(cuda_sum_aggr_arr1, cuda_sum_aggr_arr2);
    }

    double scalar_product = 0;
    SAFE_CUDA(hipMemcpy(&scalar_product, cuda_sum_aggr_arr2, sizeof(double), hipMemcpyDeviceToHost));

    double global_scalar_product = 0;

    int ret = MPI_Allreduce(
        &scalar_product,            // const void *sendbuf,
        &global_scalar_product,     // void *recvbuf,
        1,                          // int count,
        MPI_DOUBLE,                 // MPI_Datatype datatype,
        MPI_SUM,                    // MPI_Op op,
        procParams.comm             // MPI_Comm comm
    );
    if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error reducing scalar_product.");

    return global_scalar_product;
}


// ==================================================================================================================================================
//                                                                                                                     cudakernel_PrepareStopCriteria
// ==================================================================================================================================================
__global__ void cudakernel_PrepareStopCriteria (double* const cuda_sum_aggr_arr, const double* const f1, const double* const f2,
    const int arr_size){

    int threadId = THREAD_IN_GRID_ID;

    if (threadId < arr_size)
        cuda_sum_aggr_arr[threadId] = fabs(f1[threadId] - f2[threadId]);
}


// ==================================================================================================================================================
//                                                                                                                              cudakernel_ComputeMax
// ==================================================================================================================================================
// 
// This cuda kernel found maximum in `blockSize` values from function f corresponding for current block and stores the result into
//      `cuda_sum_aggr_arr[blockLinearId]`
// 
__global__ void cudakernel_ComputeMax (double* const cuda_sum_aggr_arr, const double* const f, const int arr_size){

    const int blockSize = BLOCK_SIZE;
    int thisBlockSize;
    
    extern __shared__ double data []; // shared memory in amount of 1024 doubles (1024=maxThreadsPerBlock)

    int threadLinearBlockId = THREAD_IN_BLOCK_ID;
    int blockLinearId = BLOCK_IN_GRID_ID;

    // Cut off redundant blocks
    if (blockLinearId < (arr_size -1) / blockSize +1) {

        bool lastBlock = blockLinearId == blockSize -1;
        if (lastBlock)
            thisBlockSize = arr_size % blockSize;
        else
            thisBlockSize = blockSize;

        if (not lastBlock or (lastBlock and (threadLinearBlockId < thisBlockSize)) ){

            data[threadLinearBlockId] = f[threadLinearBlockId + blockLinearId * blockSize];
            __syncthreads ();

            for (int s = 1; s < thisBlockSize; s *= 2){
                if (threadLinearBlockId % (2*s) == 0 and threadLinearBlockId + s < thisBlockSize){
                    data[threadLinearBlockId] = max(data[threadLinearBlockId], data[threadLinearBlockId + s]);
                }
                __syncthreads ();
            }

            if (threadLinearBlockId == 0)
                cuda_sum_aggr_arr[blockLinearId] = data[0];
        }
    }
}


// ==================================================================================================================================================
//                                                                                                                   DHP_PE_RA_FDM::cuda_StopCriteria
// ==================================================================================================================================================
bool DHP_PE_RA_FDM::cuda_StopCriteria(const double* const f1, const double* const f2){

    if (cuda_sum_aggr_arr1 == NULL)
        SAFE_CUDA(hipMalloc(&cuda_sum_aggr_arr1, procCoords.x_cells_num * procCoords.y_cells_num * sizeof(*cuda_sum_aggr_arr1)));
    if (cuda_sum_aggr_arr2 == NULL)
        SAFE_CUDA(hipMalloc(&cuda_sum_aggr_arr2, procCoords.x_cells_num * procCoords.y_cells_num * sizeof(*cuda_sum_aggr_arr2)));


    int dimension = procCoords.x_cells_num * procCoords.y_cells_num;
    pair<dim3, dim3> mesh = GridDistribute(dimension);
    cudakernel_PrepareStopCriteria<<<mesh.first, mesh.second>>> (cuda_sum_aggr_arr1, f1, f2, dimension);

    while (true){
        cudakernel_ComputeMax<<<mesh.first, mesh.second, devProp.maxThreadsPerBlock * sizeof(*cuda_sum_aggr_arr1)>>> (cuda_sum_aggr_arr2, cuda_sum_aggr_arr1, dimension);

        if (dimension == 1) break;
        
        dimension = mesh.second.x * mesh.second.y * mesh.second.z; // number of blocks
        mesh = GridDistribute(dimension);
        swap(cuda_sum_aggr_arr1, cuda_sum_aggr_arr2);
    }

    double norm = 0;
    SAFE_CUDA(hipMemcpy(&norm, cuda_sum_aggr_arr2, sizeof(double), hipMemcpyDeviceToHost));

    double global_norm = 0;

    int ret = MPI_Allreduce(
        &norm,                      // const void *sendbuf,
        &global_norm,               // void *recvbuf,
        1,                          // int count,
        MPI_DOUBLE,                 // MPI_Datatype datatype,
        MPI_MAX,                    // MPI_Op op,
        procParams.comm             // MPI_Comm comm
    );
    if (ret != MPI_SUCCESS) throw DHP_PE_RA_FDM_Exception("Error reducing scalar_product.");

    return global_norm < eps;
}
